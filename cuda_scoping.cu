#include "hip/hip_runtime.h"
/*
 * Author:  Hampton Ford
 * Github:  @shford
 *
 * Description:
 *  Flag framework to safely transfer data and synchronize device.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: Err no: %d, code: %s %s %d\n", code, hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


#define TARGET_COLLISIONS (5)
#define ARBITRARY_MAX_BUFF_SIZE (1024)
#define FALSE (0)
#define TRUE (1)


__device__ char* collision;
__device__ unsigned long long collision_cstring_size;
__device__ int read_collision = FALSE;
__device__ int terminate_signal = FALSE;

__global__ void kernel(unsigned long long page_locked_host_data_cstring_size)
{
    // initialize local data
    char *local_data;
    hipMalloc(&local_data, ARBITRARY_MAX_BUFF_SIZE);

    // copy global data to local
    collision_cstring_size = page_locked_host_data_cstring_size;
    hipMemcpyAsync(local_data, collision, collision_cstring_size, hipMemcpyDeviceToDevice, 0);

    // modify local data to test copying back to global
    if (collision_cstring_size + 1 < ARBITRARY_MAX_BUFF_SIZE) {
        local_data[collision_cstring_size - 1] = '8';
        local_data[collision_cstring_size] = '\0';
        ++collision_cstring_size;
    }

    // replace condition to ensure hashes match
    if (true)
    {
        // write local data to global for host polling
        hipMemcpyAsync(collision, local_data, collision_cstring_size, hipMemcpyDeviceToDevice, 0);

        // free local data
        hipFree(local_data);

        // tell host to read collision
        read_collision = TRUE;
    }

    // unsure if necessary - may be wholely unneeded or may could be deleted by applying volatile attr to collision
    // in theory do not release kernel level (L2) memory until host confirms it's been read
    int idling = 0;
    while (!terminate_signal)
    {
        ++idling; // this line is only here to keep the compiler from optimizing the busy loop away
    }
}

int main()
{
    // existing base data
    char tmp_filler_base_data[] = "1234567";
    unsigned long long page_locked_host_data_cstring_size = strlen(tmp_filler_base_data) + 1; // adjust for null terminator

    // allocate and initialize page-locked host variables
    char* page_locked_host_data;
    gpuErrchk( hipHostMalloc(&page_locked_host_data, ARBITRARY_MAX_BUFF_SIZE) );
    hipMemset(&page_locked_host_data, 0x00, sizeof(char) * ARBITRARY_MAX_BUFF_SIZE);
    strncpy_s(page_locked_host_data, page_locked_host_data_cstring_size, tmp_filler_base_data, ARBITRARY_MAX_BUFF_SIZE);

    int* collision_found;
    gpuErrchk( hipHostMalloc((void**)&collision_found, sizeof(int)) ); //todo CHECK THIS LINE w/ and w/out &
    *collision_found = FALSE;

    // allocate global mem for collision
    gpuErrchk( hipMalloc((void**)&collision, ARBITRARY_MAX_BUFF_SIZE) ); //todo CHECK THIS LINE w/ and w/out &
    gpuErrchk( hipMemset(collision, 0x00, sizeof(char) * ARBITRARY_MAX_BUFF_SIZE) );

    // copy data to collision in global device memory
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(collision), page_locked_host_data, page_locked_host_data_cstring_size, 0, hipMemcpyHostToDevice) );

    // hash until 5 hashes are found
    int collision_count = 0;
    while (collision_count != TARGET_COLLISIONS)
    {
        printf("Reached before kernel.\n");
        // execution configuration (sync device)
        kernel<<<1, 1>>>(page_locked_host_data_cstring_size);
        printf("Reached after kernel.\n");

        /*
         * ensure we're not trying to read from global memory before
         * the kernel can reserve/allocate it
         */
        int wait_cycles = 0;
        while (wait_cycles < 1000)
        {
            ++wait_cycles;
        }

        // poll collision flag
        do {
            const void* dev_N;
            size_t symbolSize;
            gpuErrchk( hipGetSymbolSize(&symbolSize, HIP_SYMBOL(read_collision)) );
            gpuErrchk( hipGetSymbolAddress((void**)&dev_N, read_collision) );
            gpuErrchk( hipMemcpy((void*)collision_found, dev_N, symbolSize, hipMemcpyDeviceToHost) );
            // read collision status from device into host flag
//            gpuErrchk( hipMemcpyFromSymbol(collision_found, HIP_SYMBOL(read_collision), sizeof(*collision_found), 0, hipMemcpyDeviceToHost) );
            printf("collision_found = %s after read.", *collision_found ? "TRUE" : "FALSE");

            if (*collision_found) {
                // read updated collision size into page_locked_host_data size
                gpuErrchk(hipMemcpyFromSymbol(&page_locked_host_data_cstring_size, HIP_SYMBOL(collision_cstring_size), sizeof(page_locked_host_data_cstring_size), 0, hipMemcpyDeviceToHost) );

                // read from collision to host mem
                gpuErrchk( hipMemcpyFromSymbol(page_locked_host_data, HIP_SYMBOL(collision), page_locked_host_data_cstring_size, 0, hipMemcpyDeviceToHost) );

                // for (int i = 0; i < 5; ++i) {
                // char byte[10];
                // hipMemcpyFromSymbol(byte, HIP_SYMBOL(collision), sizeof(char)*9);
                // int tmp = 2;
                // page_locked_host_data[i] = byte;
                // }

                // tell gpu threads to exit when they next read terminate_signal
                gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(terminate_signal), collision_found, sizeof(*collision_found), 0, hipMemcpyHostToDevice) );

                // replace condition with hash check
                if (true) // todo replace true w/ 20 bit comparisons
                {
                    // write file (note this should be multi-threaded. this is a waste of time when the gpu is likely idling)

                    // increase collision count
                    ++collision_count;

                    // reset collision_found, terminate_flag, and read_collision
                    *collision_found = FALSE;
                    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(terminate_signal), collision_found, sizeof(*collision_found), 0, hipMemcpyHostToDevice) );
                    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(read_collision), collision_found, sizeof(*collision_found), 0, hipMemcpyHostToDevice) );
                }
            }
        } while(!(*collision_found));

        // ensure kernel processes are finished
        hipDeviceSynchronize();
    }

    printf("Success... /\n");
    printf("The kernel changed the following character: %c\n", page_locked_host_data[0]);
    printf("Full modified string: %s\n", page_locked_host_data);

    hipHostFree(page_locked_host_data);
    hipHostFree(collision_found);
    hipFree((void*)collision);
}
