#include "hip/hip_runtime.h"
/*
 * Author:  Hampton Ford
 * Github:  @shford
 *
 * Description:
 *  Flag framework to safely transfer data and synchronize device
 *
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fileapi.h>


#define TARGET_COLLISIONS (5)
#define ARBITRARY_MAX_DATA_SIZE (1024)

// globally accessible device variables (L2 cache on CUDA 8.6)
__device__ volatile char* collision;
__device__ bool terminate_signal;


__global__ void kernel()
{
    while (terminate_signal == false)
    {
        // initialize local data
        char* local_data;
        hipMalloc(&local_data, ARBITRARY_MAX_DATA_SIZE);

        // copy global data to local
        hipMemcpyAsync(local_data, (const void*)collision, ARBITRARY_MAX_DATA_SIZE, hipMemcpyDeviceToDevice, 0);

        // replace this with working code
        bool hashes_match = true;
        if (hashes_match == true)
        {
            // write local data to global for host polling
            hipMemcpyAsync((void*)collision, local_data, ARBITRARY_MAX_DATA_SIZE, hipMemcpyDeviceToDevice, 0);

            // free local_data
            hipFree(local_data);

            // update global flag for host polling flag
            terminate_signal = true;
        }
    }
}

int main()
{
    // sample append data
    char* data = (char*)calloc(1, ARBITRARY_MAX_DATA_SIZE);
    sprintf_s(data, ARBITRARY_MAX_DATA_SIZE, "%s", "test");

    // flag true if collision is found
    bool collision_found = false;

    // allocate mem for collision
    hipMalloc(&collision, ARBITRARY_MAX_DATA_SIZE);

    // copy data to collision in global device memory
    hipMemcpyToSymbol(HIP_SYMBOL(&collision), data, strlen(data)+1, 0, hipMemcpyHostToDevice);

    // set terminate_signal to false in global device memory
    hipMemcpyToSymbol(HIP_SYMBOL(&terminate_signal), &collision_found, sizeof(collision_found), 0, hipMemcpyHostToDevice);

    // hash until 5 hashes are found
    int collision_count = 0;
    while (collision_count != TARGET_COLLISIONS)
    {
        // execution configuration (sync device)
        kernel<<<1, 32>>>();

        // read collision status from device todo check ptr issues in call
        hipMemcpyFromSymbol(&collision_found, HIP_SYMBOL(&terminate_signal), sizeof(collision_found), 0, hipMemcpyDeviceToHost);

        // poll collision flag
        while (collision_found != true)
        {
            // read the reported value (strlen fails on volatile - either loop or copy the whole thing)
            hipMemcpyFromSymbol(&data, HIP_SYMBOL(collision), ARBITRARY_MAX_DATA_SIZE, 0, hipMemcpyDeviceToHost);

            // check hashes match
            bool hashes_match = true;
            if (hashes_match == true)
            {
                // write file (note this should be multi-threaded. this is a waste of time when the gpu is likely idling)

                // increase collision count
                ++collision_count;
            }
        }

        // ensure kernel processes are finished
        hipDeviceSynchronize();
    }

    hipFree((void*)collision);
    free(data);

    printf("Everything worked... /");
}