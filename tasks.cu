#include "hip/hip_runtime.h"
#pragma once

#include "tasks.cuh"


__constant__ __device__ MD5_HASH d_const_md5_digest;    // store digest on L2 or L1 cache (on v8.6)
__device__ uint8_t d_collisions_found;                  // track number of collisions found by active kernel
__device__ unsigned long long d_collision_size;         // track # of characters in collision
__device__ int d_collision_flag;                        // signal host to read
__device__ unsigned long long d_hash_attempts;          // track total number of attempts per collision


__global__ void find_collisions(char* collision) {
    //===========================================================================================================
    // DECLARATIONS & INITIALIZATION
    //===========================================================================================================
    // allocate local buffer and keep track of size in case of resizing
    char* local_collision;
    unsigned long long local_collision_size = d_collision_size;
    unsigned long long local_buff_size = ARBITRARY_MAX_BUFF_SIZE;
    hipMalloc(&local_collision, local_buff_size);
    for (int byte_index = 0; byte_index <= local_collision_size; ++byte_index) {
        local_collision[byte_index] = collision[byte_index];
    }

    // allocate room for new hash
    MD5_HASH local_md5_digest;

    // allocate storage for random character
    unsigned long long random_index = 0;
    uint8_t randoms[NUM_8BIT_RANDS];

    //===========================================================================================================
    // COMPUTATIONS - GENERATE RANDS, RESIZE BUFFER, APPEND CHAR, HASH, COMPARE { EXIT }
    //===========================================================================================================
    do
    {
        ++d_hash_attempts;

        // generate a new batch of random numbers as needed
        if (random_index == NUM_8BIT_RANDS) {
            random_index = 0;
            for (int i = 0; i < NUM_32BIT_RANDS; ++i) {
                int id = threadIdx.x + blockIdx.x * blockDim.x;
                hiprandStatePhilox4_32_10_t state;
                hiprand_init(i, id, 0, &state);
                // assign 4 bytes at a time
                randoms[i*4] = hiprand(&state);
            }
        }
        ++random_index;

        // resize local_collision
        if (local_collision_size == ARBITRARY_MAX_BUFF_SIZE) {
            // retain ptr to old buffer
            char* old_buff = local_collision;

            // reassign local_collision ptr to new buffer
            local_buff_size *= 2;
            hipMalloc(&local_collision, local_buff_size);

            // copy data from old buffer to new buffer
            for (int i = 0; i < ARBITRARY_MAX_BUFF_SIZE; ++i) {
                local_collision[i] = old_buff[i];
            }

            // free original buffer
            hipFree(old_buff);
        }

        // append random char
        uint8_t character = randoms[random_index];
        local_collision[local_collision_size - 1] = character ? character : 1; // no premature null terminators
        local_collision[local_collision_size] = '\0';
        ++local_collision_size;

        // generate new hash
        Md5Calculate((const void*)local_collision, local_collision_size, &local_md5_digest);

        // terminate all threads if first 20 bits of digest match
        if ( ((uint32_t)*d_const_md5_digest.bytes >> 12) == ((uint32_t)*local_md5_digest.bytes >> 12))
        {
            /* todo:
             *  unlikely but possible device wide deadlock if within the same warp
             *  1 thread sets a mutex causing a divergent instruction path and the
             *  scheduler interrupts said thread to schedule another which will then idle
             *  forever, thus preventing the mutex thread from completing.
             *  May want to utilize capability: " Run time limit on kernels:                     Yes"
             */

            // wait for resources to be released before waiting
            while (d_collision_flag) {
                // idle
            }

            // set synchronization barrier/mutex on d_collision_flag, d_collision_size, collision


            // write local_data, local_data_size to global for host polling
            for (int byte_index = 0; byte_index <= local_collision_size; ++byte_index) {
                collision[byte_index] = local_collision[byte_index];
            }

            // tell host to read collision
            d_collision_flag = TRUE;

            while (d_collision_flag) {
                // release synchronization barrier/mutex once host has reading and resets flag
            }
        }
    } while(d_collisions_found < TARGET_COLLISIONS);

}

void task1() {
    //===========================================================================================================
    // SEQUENTIAL TASKS (Initial)
    //===========================================================================================================

    // todo v5 hipHostMalloc - code chunk has been tested
    // char* h_page_locked_data;
    // gpuErrchk( hipHostMalloc(&h_page_locked_data, ARBITRARY_MAX_BUFF_SIZE) );
    // hipMemset(&h_page_locked_data, 0x00, sizeof(char) * ARBITRARY_MAX_BUFF_SIZE);

    // read file data
    char sampleFile_path[] = "C:/Users/shford/CLionProjects/cuda_hashing/sample.txt";
    char* h_sampleFile_buff;
    uint32_t h_sampleFile_buff_size = 0; // handle files up to ~4GiB (2^32-1 bytes)
    get_file_data((char*)sampleFile_path, &h_sampleFile_buff, &h_sampleFile_buff_size);

    // get hash md5_digest
    MD5_HASH md5_digest;
    Md5Calculate((const void*)h_sampleFile_buff, h_sampleFile_buff_size, &md5_digest);

    // format and print digest as a string of hex characters
    char hash[MD5_HASH_SIZE_B + 1]; //MD5 len is 16B, 1B = 2 chars

    char tiny_hash[TINY_HASH_SIZE_B + 1];
    for (int i = 0; i < MD5_HASH_SIZE / 2; ++i)
    {
        sprintf(hash + i * 2, "%2.2x", md5_digest.bytes[i]);
    }
    hash[sizeof(hash)-1] = '\0';
    strncpy_s(tiny_hash, sizeof(tiny_hash), hash, _TRUNCATE);
    tiny_hash[sizeof(tiny_hash)-1] = '\0';

    printf("Full MD5 md5_digest is: %s\n", hash);
    printf("TinyHash md5_digest is: %s\n\n", tiny_hash);

    //===========================================================================================================
    // BEGIN CUDA PARALLELIZATION
    //===========================================================================================================

    // allocate storage for collisions once found
    char* h_collisions[TARGET_COLLISIONS];
    unsigned long long h_collision_sizes[TARGET_COLLISIONS];
    unsigned long long h_collision_attempts = 0;
    for (int i = 0; i < TARGET_COLLISIONS; ++i) {
        h_collisions[i] = (char*)calloc(1, ARBITRARY_MAX_BUFF_SIZE);
        h_collision_sizes[i] = 0;
    }
    uint8_t h_num_collisions_found = 0;
    int h_collision_flag = FALSE;

    // allocate global mem for collision - initialized in loop
    char* d_collision;
    gpuErrchk( hipMalloc((void **)&d_collision, ARBITRARY_MAX_BUFF_SIZE) );

    // parallelization setup
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_collisions_found), &h_num_collisions_found, sizeof(h_num_collisions_found), 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_collision_flag), &h_collision_flag, sizeof(h_collision_flag), 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_const_md5_digest), &md5_digest, sizeof(md5_digest), 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_collision_size), &h_sampleFile_buff_size, sizeof(h_sampleFile_buff_size), 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_collision, h_sampleFile_buff, h_sampleFile_buff_size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_hash_attempts), &h_collision_attempts, sizeof(h_collision_attempts), 0, hipMemcpyHostToDevice) );

    // run kernel
    while (hipMemcpyFromSymbol(&h_num_collisions_found, HIP_SYMBOL(d_collisions_found), sizeof(h_num_collisions_found), 0, hipMemcpyDeviceToHost) == hipSuccess && h_num_collisions_found < TARGET_COLLISIONS)
    {
        // execution configuration (sync device)
        find_collisions<<<MULTIPROCESSORS, CUDA_CORES_PER_MULTIPROCESSOR>>>(d_collision);

        // poll collision flag
        while (!h_collision_flag)
        {
            // read collision status from device into host flag
            gpuErrchk( hipMemcpyFromSymbol(&h_collision_flag, HIP_SYMBOL(d_collision_flag), sizeof(h_collision_flag), 0, hipMemcpyDeviceToHost) );

            if (h_collision_flag)
            {
                // read updated collision count, collision size, collision, and hash attempts
                gpuErrchk(hipMemcpyFromSymbol(&h_num_collisions_found, HIP_SYMBOL(d_collisions_found), sizeof(h_num_collisions_found), 0, hipMemcpyDeviceToHost) );
                gpuErrchk(hipMemcpyFromSymbol(&h_collision_sizes[h_num_collisions_found], HIP_SYMBOL(d_collision_size), sizeof(h_sampleFile_buff_size), 0, hipMemcpyDeviceToHost) );
                gpuErrchk(hipMemcpy(h_collisions[h_num_collisions_found], d_collision, h_sampleFile_buff_size, hipMemcpyDeviceToHost) );
                gpuErrchk(hipMemcpyFromSymbol(&h_collision_attempts, HIP_SYMBOL(h_collision_attempts),
                                               sizeof(h_collision_attempts), 0, hipMemcpyDeviceToHost) );

                // reset flag to continue
                h_collision_flag = FALSE;
                hipMemcpyToSymbol(HIP_SYMBOL(d_collision_flag), &h_collision_flag, sizeof(h_collision_flag), 0, hipMemcpyHostToDevice);

            }
        }
    }
    gpuErrchk( hipDeviceSynchronize() );

    // error handling
    if (h_num_collisions_found < TARGET_COLLISIONS)
    {
        printf("Error: Failed to update the number of collisions from h_num_collisions_found.\n");
    }
    else
    {
        printf("\nCalculated %d collisions... Success/\n", TARGET_COLLISIONS);
    }

    // free collisions
    hipHostFree(h_sampleFile_buff);
    hipFree(d_collision);
    free((void*)h_sampleFile_buff);

    //===========================================================================================================
    // WRITE COLLISIONS TO DISK
    //===========================================================================================================

    printf("Original string: %s\n", h_sampleFile_buff);
    for (int i = 0; i < TARGET_COLLISIONS; ++i) {
        printf("Collision %d: %s\n", i, h_collisions[i]);

        // todo write collision

        // free collision once written
        free(h_collisions[i]);
    }
}
